/* This work is licensed under the terms of the GNU GPL, version 2.  See
 * the GPL-COPYING file in the top-level directory.
 *
 * Copyright (c) 2010-2011 University of Utah and the Flux Group.
 * All rights reserved.
 */
 
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../../../kgpu/kgpu.h"
#include "../../../kgpu/gputils.h"
#include "../gaesu.h"

#define BYTES_PER_BLOCK  1024
#define BYTES_PER_THREAD 4
#define BYTES_PER_GROUP  16
#define THREAD_PER_BLOCK (BYTES_PER_BLOCK/BYTES_PER_THREAD)
#define WORDS_PER_BLOCK (BYTES_PER_BLOCK/4)

#define BPT_BYTES_PER_BLOCK 4096

struct kgpu_service gaes_ecb_enc_srv;
struct kgpu_service gaes_ecb_dec_srv;

struct kgpu_service gaes_ctr_srv;
struct kgpu_service gaes_lctr_srv;

struct kgpu_service bp4t_gaes_ecb_enc_srv;
struct kgpu_service bp4t_gaes_ecb_dec_srv;

struct kgpu_service gaes_xts_enc_srv;
struct kgpu_service gaes_xts_dec_srv;

struct gaes_ecb_data {
    u32 *d_key;
    u32 *h_key;
    int nrounds;
    int nr_dblks_per_tblk;
};

struct gaes_ctr_data {
    u32 *d_key;
    u32 *h_key;
    u8 *d_ctr;
    u8 *h_ctr;
    int nrounds;
    int nr_dblks_per_tblk;
};

#if 0
static void dump_hex(u8* p, int rs, int cs)
{
    int r,c;
    printf("\n");
    for (r=0; r<rs; r++) {
	for (c=0; c<cs; c++) {
	    printf("%02x ", p[r*cs+c]);
	}
	printf("\n");
    }
}
#endif /* test only */

/*
 * Include device code
 */
#include "dev.cu"
#include "aes.cu"
#include "setupAES.cpp"

int gaes_ecb_compute_size_bpt(struct kgpu_service_request *sr)
{
    sr->block_x =
	sr->outsize>=BPT_BYTES_PER_BLOCK?
	BPT_BYTES_PER_BLOCK/16: sr->outsize/16;
    sr->grid_x =
	sr->outsize/BPT_BYTES_PER_BLOCK?
	sr->outsize/BPT_BYTES_PER_BLOCK:1;
    sr->block_y = 1;
    sr->grid_y = 1;

    return 0;
}

int gaes_ecb_compute_size_bp4t(struct kgpu_service_request *sr)
{
    sr->block_y =
	sr->outsize>=BYTES_PER_BLOCK?
	BYTES_PER_BLOCK/BYTES_PER_GROUP: (sr->outsize/BYTES_PER_GROUP);
    sr->grid_x =
	sr->outsize/BYTES_PER_BLOCK?
	sr->outsize/BYTES_PER_BLOCK:1;
    sr->block_x = BYTES_PER_GROUP/BYTES_PER_THREAD;
    sr->grid_y = 1;

    return 0;
}

#define RKLENGTH(keybits) ((keybits)/8+28) 
static int count = 0;
int gaes_ecb_launch_bpt(struct kgpu_service_request *sr)
{
    //struct crypto_aes_ctx *hctx = (struct crypto_aes_ctx*)sr->hdata;
    //struct crypto_aes_ctx *dctx = (struct crypto_aes_ctx*)sr->ddata;
	u8 *hkey;
	u8 *dkey;
	int blockDimX = 256+256;
	int gridDimX = (sr->outsize/16+blockDimX)/blockDimX;
	int keybits = 256;
    int i;
	//u32 *hencrypt_rk;
	//u32 *hdecrypt_rk;
	//u32 htd0[256];
	//u32 *dtd0;

	//hencrypt_rk = (u32 *)malloc(RKLENGTH(keybits)*sizeof(u32));
	//hdecrypt_rk = (u32 *)malloc(RKLENGTH(keybits)*sizeof(u32));

    //hipMalloc(&dtd0, 256*sizeof(u32));
    //hipMemcpy(dtd0, htd0, 256*sizeof(u32), hipMemcpyHostToDevice);

	dkey = (u8 *)sr->ddata;
	hkey = (u8 *)sr->hdata;
    //memcpy(hkey,"01234567890123456789012345678912",32);
    //printf("hkey= %s \n",hkey);



    u32 *rk = new u32[RKLENGTH(keybits)];

    nrounds_en = rijndaelSetupEncrypt(rk, hkey, keybits);

	/*
    printf("call setpAES_cuda--en--rk_len: %d\n", RKLENGTH(keybits));
    for (i=0; i<60; i++) {
        //rk[i] = entmp[i];
        printf("%lu\n", rk[i]);
    }    
	*/

    hipMalloc(&encrypt_rk_dev, RKLENGTH(keybits)*sizeof(u32));
    hipMemcpy(encrypt_rk_dev, rk, RKLENGTH(keybits)*sizeof(u32), hipMemcpyHostToDevice);

    nrounds_de = rijndaelSetupDecrypt(rk, hkey, keybits);
	/*
    printf("call setpAES_cuda--de--rk_len: %d\n", RKLENGTH(keybits));
    for (i=0; i<60; i++) {
        //rk[i] = detmp[i];
        printf("%lu\n", rk[i]);
    }
	*/
    hipMalloc(&decrypt_rk_dev, RKLENGTH(keybits)*sizeof(u32));
    hipMemcpy(decrypt_rk_dev, rk, RKLENGTH(keybits)*sizeof(u32), hipMemcpyHostToDevice);


    if (sr->s == &gaes_ecb_dec_srv) {
    decryptKernel
		<<<gridDimX,blockDimX,
	    0, (hipStream_t)(sr->stream)>>>
	    (
		decrypt_rk_dev,
		nrounds_de,
		(u8*)sr->dout,
		(u8*)sr->dout,
		(int)sr->outsize/16
		);
		/*
    	hipMemcpy(htd0, dtd0, 256*sizeof(u32), hipMemcpyDeviceToHost);
		printf("debug:=============htd0=====:\n");
    	for (i=0; i<256; i++) {
        	printf("%lu\n", htd0[i]);
    	}
		*/
	}
    else {
	encryptKernel
		<<<gridDimX,blockDimX,
	    0, (hipStream_t)(sr->stream)>>>
	    (
		encrypt_rk_dev,
		nrounds_en,
		(u8*)sr->dout,
		(u8*)sr->dout,
		(int)sr->outsize/16
		);
		/*
    	hipMemcpy(hencrypt_rk, encrypt_rk_dev, RKLENGTH(keybits)*sizeof(u32), hipMemcpyDeviceToHost);
		printf("hencrypt_rk:\n");
    	for (i=0; i<60; i++) {
        	printf("%lu\n", hencrypt_rk[i]);
    	}    
		*/
	}
	hipFree(decrypt_rk_dev);
	hipFree(encrypt_rk_dev);
    return 0;
}

int gaes_ecb_launch_bp4t(struct kgpu_service_request *sr)
{
    struct crypto_aes_ctx *hctx = (struct crypto_aes_ctx*)sr->hdata;
    struct crypto_aes_ctx *dctx = (struct crypto_aes_ctx*)sr->ddata;
    
    if (sr->s == &gaes_ecb_dec_srv)        
	aes_decrypt_bp4t<<<
	    dim3(sr->grid_x, sr->grid_y),
	    dim3(sr->block_x, sr->block_y),
	    0, (hipStream_t)(sr->stream)>>>
	    ((u32*)dctx->key_dec,
	     hctx->key_length/4+6,
	     (u8*)sr->dout);
    else
	aes_encrypt_bp4t<<<
	    dim3(sr->grid_x, sr->grid_y),
	    dim3(sr->block_x, sr->block_y),
	    0, (hipStream_t)(sr->stream)>>>
	    ((u32*)dctx->key_enc,
	     hctx->key_length/4+6,
	     (u8*)sr->dout);
   
    return 0;
}

int gaes_ecb_prepare(struct kgpu_service_request *sr)
{
    hipStream_t s = (hipStream_t)(sr->stream);//gpu_get_stream(sr->stream_id);
    
    csc( ah2dcpy( sr->din, sr->hin, sr->insize, s) );
    
    return 0;
}

int gaes_ecb_post(struct kgpu_service_request *sr)
{
    hipStream_t s = (hipStream_t)(sr->stream);//gpu_get_stream(sr->stream_id);

    csc( ad2hcpy( sr->hout, sr->dout, sr->outsize, s) );
    
    return 0;
}

#define gaes_xts_post gaes_ecb_post
#define gaes_xts_prepare gaes_ecb_prepare

int gaes_xts_compute_size(struct kgpu_service_request *sr)
{
    sr->block_x = XTS_SECTOR_SIZE/AES_BLOCK_SIZE;
    sr->grid_x = sr->outsize/XTS_SECTOR_SIZE;
    sr->block_y = 1;
    sr->grid_y = 1;

    return 0;
}

int gaes_xts_launch(struct kgpu_service_request *sr)
{
    struct crypto_xts_info *hinfo =
	(struct crypto_xts_info*)(sr->hdata);
    struct crypto_xts_info *dinfo =
	(struct crypto_xts_info*)(sr->ddata);

    if (sr->s == &gaes_xts_dec_srv) 
        xts_decrypt<<<
	dim3(sr->grid_x, sr->grid_y),
	dim3(sr->block_x, sr->block_y),
	0, (hipStream_t)(sr->stream)>>>
	    ((u32*)dinfo->key_dec,
	     hinfo->key_length/4+6,
	     (u8*)sr->dout,
	     dinfo->ivs);
    else
        xts_encrypt<<<
	dim3(sr->grid_x, sr->grid_y),
	dim3(sr->block_x, sr->block_y),
	0, (hipStream_t)(sr->stream)>>>
	    ((u32*)dinfo->key_enc,
	     hinfo->key_length/4+6,
	     (u8*)sr->dout,
	     dinfo->ivs);
    return 0;
}

#define gaes_ctr_compute_size gaes_ecb_compute_size_bpt
#define gaes_ctr_post gaes_ecb_post
#define gaes_ctr_prepare gaes_ecb_prepare

int gaes_lctr_compute_size(struct kgpu_service_request *sr)
{
    struct crypto_gaes_ctr_info *info
	= (struct crypto_gaes_ctr_info*)(sr->hdata);
    sr->block_x = info->ctr_range/16;
    sr->grid_x = sr->outsize/sr->block_x;
    sr->block_y = 1;
    sr->grid_y = 1;

    return 0;
}

int gaes_ctr_launch(struct kgpu_service_request *sr)
{
    struct crypto_gaes_ctr_info *hinfo =
	(struct crypto_gaes_ctr_info*)(sr->hdata);
    struct crypto_gaes_ctr_info *dinfo =
	(struct crypto_gaes_ctr_info*)(sr->ddata);

    aes_ctr_crypt<<<
	dim3(sr->grid_x, sr->grid_y),
	dim3(sr->block_x, sr->block_y),
	0, (hipStream_t)(sr->stream)>>>
	((u32*)dinfo->key_enc,
	 hinfo->key_length/4+6,
	 (u8*)sr->dout,
	 dinfo->ctrblk);
    return 0;
}

int gaes_lctr_launch(struct kgpu_service_request *sr)
{
    struct crypto_gaes_ctr_info *hinfo =
	(struct crypto_gaes_ctr_info*)(sr->hdata);
    struct crypto_gaes_ctr_info *dinfo =
	(struct crypto_gaes_ctr_info*)(sr->ddata);
    
    aes_lctr_crypt<<<
	dim3(sr->grid_x, sr->grid_y),
	dim3(sr->block_x, sr->block_y),
	0, (hipStream_t)(sr->stream)>>>
	((u32*)dinfo->key_enc,
	 hinfo->key_length/4+6,
	 (u8*)sr->dout,
	 dinfo->ctrblk);
    return 0;
}

/*
 * Naming convention of ciphers:
 * g{algorithm}_{mode}[-({enc}|{dev})]
 *
 * {}  : var value
 * []  : optional
 * (|) : or
 */
extern "C" int init_service(void *lh, int (*reg_srv)(struct kgpu_service*, void*))
{
    int err;
    printf("[libsrv_gaes] Info: init gaes services\n");

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(decryptKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(encryptKernel), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(aes_decrypt_bp4t), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(aes_encrypt_bp4t), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(aes_ctr_crypt), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(aes_lctr_crypt), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(xts_decrypt), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(xts_encrypt), hipFuncCachePreferL1);
    
    sprintf(gaes_ecb_enc_srv.name, "gaes_ecb-enc");
    gaes_ecb_enc_srv.sid = 0;
    gaes_ecb_enc_srv.compute_size = gaes_ecb_compute_size_bpt;
    gaes_ecb_enc_srv.launch = gaes_ecb_launch_bpt;
    gaes_ecb_enc_srv.prepare = gaes_ecb_prepare;
    gaes_ecb_enc_srv.post = gaes_ecb_post;
    
    sprintf(gaes_ecb_dec_srv.name, "gaes_ecb-dec");
    gaes_ecb_dec_srv.sid = 0;
    gaes_ecb_dec_srv.compute_size = gaes_ecb_compute_size_bpt;
    gaes_ecb_dec_srv.launch = gaes_ecb_launch_bpt;
    gaes_ecb_dec_srv.prepare = gaes_ecb_prepare;
    gaes_ecb_dec_srv.post = gaes_ecb_post;

    sprintf(gaes_ctr_srv.name, "gaes_ctr");
    gaes_ctr_srv.sid = 0;
    gaes_ctr_srv.compute_size = gaes_ctr_compute_size;
    gaes_ctr_srv.launch = gaes_ctr_launch;
    gaes_ctr_srv.prepare = gaes_ctr_prepare;
    gaes_ctr_srv.post = gaes_ctr_post;

    sprintf(gaes_lctr_srv.name, "gaes_lctr");
    gaes_lctr_srv.sid = 0;
    gaes_lctr_srv.compute_size = gaes_lctr_compute_size;
    gaes_lctr_srv.launch = gaes_lctr_launch;
    gaes_lctr_srv.prepare = gaes_ctr_prepare;
    gaes_lctr_srv.post = gaes_ctr_post;
    
    sprintf(gaes_xts_enc_srv.name, "gaes_xts-enc");
    gaes_xts_enc_srv.sid = 0;
    gaes_xts_enc_srv.compute_size = gaes_xts_compute_size;
    gaes_xts_enc_srv.launch = gaes_xts_launch;
    gaes_xts_enc_srv.prepare = gaes_xts_prepare;
    gaes_xts_enc_srv.post = gaes_xts_post;
    
    sprintf(gaes_xts_dec_srv.name, "gaes_xts-dec");
    gaes_xts_dec_srv.sid = 0;
    gaes_xts_dec_srv.compute_size = gaes_xts_compute_size;
    gaes_xts_dec_srv.launch = gaes_xts_launch;
    gaes_xts_dec_srv.prepare = gaes_xts_prepare;
    gaes_xts_dec_srv.post = gaes_xts_post;

    err = reg_srv(&gaes_ecb_enc_srv, lh);
    err |= reg_srv(&gaes_ecb_dec_srv, lh);
    err |= reg_srv(&gaes_ctr_srv, lh);
    err |= reg_srv(&gaes_lctr_srv, lh);
    err |= reg_srv(&gaes_xts_enc_srv, lh);
    err |= reg_srv(&gaes_xts_dec_srv, lh);
    if (err) {
    	fprintf(stderr,
		"[libsrv_gaes] Error: failed to register gaes services\n");
    } 
    
    return err;
}

extern "C" int finit_service(void *lh, int (*unreg_srv)(const char*))
{
    int err;
    printf("[libsrv_gaes] Info: finit gaes services\n");
    
    err = unreg_srv(gaes_ecb_enc_srv.name);
    err |= unreg_srv(gaes_ecb_dec_srv.name);
    err |= unreg_srv(gaes_ctr_srv.name);
    err |= unreg_srv(gaes_lctr_srv.name);
    err |= unreg_srv(gaes_xts_enc_srv.name);
    err |= unreg_srv(gaes_xts_dec_srv.name);
    if (err) {
    	fprintf(stderr,
		"[libsrv_gaes] Error: failed to unregister gaes services\n");
    }
    
    return err;
}


