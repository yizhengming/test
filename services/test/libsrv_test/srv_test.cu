#include "hip/hip_runtime.h"
/* This work is licensed under the terms of the GNU GPL, version 2.  See
 * the GPL-COPYING file in the top-level directory.
 *
 * Copyright (c) 2010-2011 University of Utah and the Flux Group.
 * All rights reserved.
 */
 
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../../../kgpu/kgpu.h"
#include "../../../kgpu/gputils.h"

__global__ void empty_kernel(void)
{
}

static int empty_cs(struct kgpu_service_request *sr)
{
    sr->block_x = 1;
    sr->grid_x = 1;
    sr->block_y = 1;
    sr->grid_y = 1;
    return 0;
}

static int empty_launch(struct kgpu_service_request *sr)
{
    empty_kernel<<<dim3(sr->grid_x, sr->grid_y),
	dim3(sr->block_x, sr->block_y), 0, (hipStream_t)(sr->stream)>>>();
    return 0;
}

static int empty_prepare(struct kgpu_service_request *sr)
{
    hipStream_t s = (hipStream_t)(sr->stream);
    csc( ah2dcpy( sr->din, sr->hin, sr->insize, s) );
    return 0;
}

static int empty_post(struct kgpu_service_request *sr)
{
    hipStream_t s = (hipStream_t)(sr->stream);
    csc( ad2hcpy( sr->hout, sr->dout, sr->outsize, s) );
    return 0;
}

static struct kgpu_service empty_srv;

extern "C" int init_service(void *lh, int (*reg_srv)(struct kgpu_service*, void*))
{
    printf("[libsrv_test] Info: init test service\n");
    
    sprintf(empty_srv.name, "empty_service");
    empty_srv.sid = 1;
    empty_srv.compute_size = empty_cs;
    empty_srv.launch = empty_launch;
    empty_srv.prepare = empty_prepare;
    empty_srv.post = empty_post;

    return reg_srv(&empty_srv, lh);
}

extern "C" int finit_service(void *lh, int (*unreg_srv)(const char*))
{
    printf("[libsrv_test] Info: finit test service\n");
    return unreg_srv(empty_srv.name);
}
